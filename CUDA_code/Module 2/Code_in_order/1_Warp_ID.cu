#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""

__global__ void print_details_of_warps()
{
	int gid = blockIdx.y * gridDim.x * blockDim.x		//Y-offset
			  + blockIdx.x * blockDim.x + threadIdx.x;	//X-offset & actual Thread Idx

	int warp_id = threadIdx.x / 32;	//Lets' see this one | threadIdx.x / 32

	int gbid = blockIdx.y * gridDim.x + blockIdx.x;	//Block Index of y * Grid dim X (2) + block Index | Offset Y Block + Block X Idx

	printf("tid : %d, bid.x : %d, bid.y : %d, gid : %d, warp_id : %d, gbid : %d \n",
			threadIdx.x, blockIdx.x, blockIdx.y, gid, warp_id, gbid);
}

int main(int argc , char** argv)
{
	dim3 block_size(42);
	dim3 grid_size(2,2);

	print_details_of_warps << <grid_size,block_size >> > ();
	hipDeviceSynchronize();

	hipDeviceReset();
	return EXIT_SUCCESS;
}