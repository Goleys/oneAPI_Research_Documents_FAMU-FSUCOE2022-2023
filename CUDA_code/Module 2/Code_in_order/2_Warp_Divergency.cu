#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuda_common.cuh"

__global__ void code_without_divergence()
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;    //X dimensional Grid

    float a, b;
    a = b = 0;

    int warp_id = gid / 32; //Calculate Warp Id by dividing GID by 32 (remember that a warp has 32 threads inside)

    if (warp_id % 2 == 0)
    {
        a = 100.0;
        b = 50.0;
    }
    else
    {
        a = 200.0;
        b = 75.0;
    }
}

__global__ void divergence_code()
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;    //X dimensional Grid

    float a, b;
    a = b = 0;

    //When calculating branch efficiency, it will take the portions of code inside both if and else blocks
    //In consideration. Also, the CUDA compiler will still optimize some aspects of our code regardless of 
	// us turning off the compiler
    if (gid % 2 == 0)   //Assigning Values based on Thread Id's   
    {
        a = 100.0;
        b = 50.0;
    }
    else
    {
        a = 200.0;
        b = 75.0;
    }
}

int main(int argc, char** argv)
{
    printf("\n-----------------------WARP DIVERGENCE EXAMPLE------------------------ \n\n");

    int size = 1 << 22;

    dim3 block_size(128);
    dim3 grid_size((size + block_size.x - 1) / block_size.x);

    code_without_divergence << <grid_size, block_size >> > ();
    hipDeviceSynchronize();

    divergence_code << <grid_size, block_size >> > ();
    hipDeviceSynchronize();

    hipDeviceReset();
    return 0;
}