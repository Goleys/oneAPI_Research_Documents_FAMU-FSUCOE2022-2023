#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "cuda_common.cuh"
#include <stdio.h>

//for random intialize
#include <stdlib.h>
#include <time.h>

//for memset
#include <cstring>

__global__ void sum_arrays_gpu(int* a, int* b, int* c, int size)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;

	if (index < size)
		c[index] = a[index] + b[index];
}

//compare two arrays
void compare_arrays(int* a, int* b, int size)
{
	bool eq = true;
	for (int i = 0; i < size; i++)
	{
		if (a[i] != b[i])
		{
			bool eq = false;
		}
	}
	if (eq == false)
		printf("Arrays are different \n");
	else
		printf("Arrays are the same \n");
}

void sum_arrays_cpu(int* a, int* b, int* c, int size)
{
	for (int i = 0; i < size; i++)
	{
		c[i] = a[i] + b[i];
	}
}

int main()
{
	int size = 1 << 25;
	int block_size = 128;
	hipError_t error;	//We assign the variable to the functions we used

	//number of bytes needed to hold element count
	int n_bytes = size * sizeof(int);	//Same as NO_bytes

	//Host Pointer
	int* h_a, * h_b, * gpu_results, * h_c;	//Add another pointer to hold array c

	//allocate memory for host size pointers
	h_a = (int*)malloc(n_bytes);
	h_b = (int*)malloc(n_bytes);
	gpu_results = (int*)malloc(n_bytes);
	h_c = (int*)malloc(n_bytes);

	//initialize h_a and h_b arrays randomly
	time_t t;
	srand((unsigned)time(&t));

	for (size_t i = 0; i < size; i++)
		h_a[i] = (int)(rand() & 0xFF);

	for (size_t i = 0; i < size; i++)
		h_b[i] = (int)(rand() & 0xFF);

	//Calculate array c in CPU
	sum_arrays_cpu(h_a, h_b, h_c, size);

	//Set gpu_results in memory
	memset(gpu_results, 0, n_bytes);

	//Device Pointers
	int* d_a, * d_b, * d_c;

	//allocate memory for device size pointers
	gpuErrchk(hipMalloc((int**)&d_a, n_bytes));
	gpuErrchk(hipMalloc((int**)&d_b, n_bytes));
	gpuErrchk(hipMalloc((int**)&d_c, n_bytes));

	//Memory transfer from Host to Device
	hipMemcpy(d_a, h_a, n_bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, n_bytes, hipMemcpyHostToDevice);

	//Launching the grid
	dim3 block(block_size);
	dim3 grid((size / block.x) + 1);

	sum_arrays_gpu << < grid, block >> > (d_a, d_b, d_c, size);
	hipDeviceSynchronize();

	//Memory Tranfer back to host 
	hipMemcpy(d_c, gpu_results, n_bytes, hipMemcpyDeviceToHost);

	//array comparison
	compare_arrays(gpu_results, h_c, size);

	//We then reclaim the memory from host and device
	hipFree(d_c); hipFree(d_b); hipFree(d_a);
	free(h_a); free(h_b); free(gpu_results);
}

